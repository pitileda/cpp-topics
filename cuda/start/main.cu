
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void test_kernel(int *data) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  data[idx] = idx;
}

int main() {
  int *d_data;
  hipMalloc(&d_data, 8 * sizeof(int));

  test_kernel<<<2, 4>>>(d_data);

  hipError_t launch_err = hipGetLastError();
  printf("Kernel launch: %s\n", hipGetErrorString(launch_err));

  hipError_t sync_err = hipDeviceSynchronize();
  printf("Kernel sync: %s\n", hipGetErrorString(sync_err));

  hipFree(d_data);
  return 0;
}
