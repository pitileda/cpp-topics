
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void dummy_kernel(int *out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  out[idx] = idx;
}

int main() {
  const int N = 8;
  int h_out[N] = {0};
  int *d_out;

  hipMalloc(&d_out, N * sizeof(int));
  dummy_kernel<<<2, 4>>>(d_out);

  hipError_t launch_err = hipGetLastError();
  if (launch_err != hipSuccess) {
    printf("Kernel launch error: %s\n", hipGetErrorString(launch_err));
    return 1;
  }

  hipError_t sync_err = hipDeviceSynchronize();
  if (sync_err != hipSuccess) {
    printf("Kernel sync error: %s\n", hipGetErrorString(sync_err));
    return 1;
  }

  hipMemcpy(h_out, d_out, N * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; ++i) printf("h_out[%d] = %d\n", i, h_out[i]);

  hipFree(d_out);
  return 0;
}
